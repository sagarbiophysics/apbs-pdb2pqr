#include "hip/hip_runtime.h"
/**
 *  @ingroup PMGC
 *  @author  Tucker Beck [fortran ->c translation], Michael Holst [original]
 *  @brief
 *  @version $Id:
 *
 *  @attention
 *  @verbatim
 *
 * APBS -- Adaptive Poisson-Boltzmann Solver
 *
 * Nathan A. Baker (nathan.baker@pnl.gov)
 * Pacific Northwest National Laboratory
 *
 * Additional contributing authors listed in the code documentation.
 *
 * Copyright (c) 2010-2014 Battelle Memorial Institute. Developed at the Pacific Northwest National Laboratory, operated by Battelle Memorial Institute, Pacific Northwest Division for the U.S. Department Energy.  Portions Copyright (c) 2002-2010, Washington University in St. Louis.  Portions Copyright (c) 2002-2010, Nathan A. Baker.  Portions Copyright (c) 1999-2002, The Regents of the University of California. Portions Copyright (c) 1995, Michael Holst.
 * All rights reserved.
 *
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * -  Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * - Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * - Neither the name of Washington University in St. Louis nor the names of its
 * contributors may be used to endorse or promote products derived from this
 * software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
 * LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * @endverbatim
 */

#include "gsd.hu"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>

#define HANDLE_ERROR(x){									\
	hipError_t _err = x;									\
	if(_err != hipSuccess){								\
		printf("(%s:%d)Cuda error: %s\n", __FILE__, __LINE__, hipGetErrorString(_err));	\
		exit(-1);											\
	}														\
}

__global__ void cuTest(float *x, float *x2, float *fc, float *cc, float *oC, float *uC, float *oE, float *oN, int N, int dx, int dy, int dz){

	int ind = blockDim.x*blockIdx.x + threadIdx.x;
	
	int lb = (dx*dy) + dx + 1;
	int ub = (dz-2)*(dx*dy)+(dy-2)*dx+(dx-2);
	
	if(ind >= lb && ind <= ub && oC[ind] != 0){
		x2[ind] = ( (fc[ind] 
			+ oN[ind]	* x[ind+dx] 
			+ oN[ind-dx]	* x[ind-dx] 
			+ oE[ind]	* x[ind+1] 
			+ oE[ind-1]	* x[ind-1]
			+ uC[ind]	* x[ind+dx*dy] 
			+ uC[ind-dx*dy]	* x[ind-dx*dy] ) 
			/ oC[ind] ) 
			+ cc[ind];
	}
}

VPUBLIC void Vgsrb(int *nx, int *ny, int *nz,
        int *ipc, double *rpc,
        double *ac, double *cc, double *fc,
        double *x, double *w1, double *w2, double *r,
        int *itmax, int *iters,
        double *errtol, double *omega,
        int *iresid, int *iadjoint) {

    int numdia; /// @todo: doc

    MAT2(ac, *nx * *ny * *nz, 1);

    // Do in one step ***
    numdia = VAT(ipc, 11);
    if (numdia == 7) {
        Vgsrb7x(nx, ny, nz,
                ipc, rpc,
                RAT2(ac, 1,1), cc, fc,
                RAT2(ac, 1,2), RAT2(ac, 1,3), RAT2(ac, 1,4),
                x, w1, w2, r,
                itmax, iters, errtol, omega, iresid, iadjoint);
    } else if (numdia == 27) {
        Vgsrb27x(nx, ny, nz,
                 ipc, rpc,
                 RAT2(ac, 1, 1), cc, fc,
                 RAT2(ac, 1, 2), RAT2(ac, 1, 3), RAT2(ac, 1, 4),
                 RAT2(ac, 1, 5), RAT2(ac, 1, 6),
                 RAT2(ac, 1, 7), RAT2(ac, 1, 8), RAT2(ac, 1, 9), RAT2(ac, 1,10),
                 RAT2(ac, 1,11), RAT2(ac, 1,12), RAT2(ac, 1,13), RAT2(ac, 1,14),
                 x, w1, w2, r,
                 itmax, iters, errtol, omega, iresid, iadjoint);
    } else {
        Vnm_print(2, "GSRB: invalid stencil type given...\n");
    }
}



VPUBLIC void Vgsrb7x(int *nx,int *ny,int *nz,
        int *ipc, double *rpc,
        double *oC, double *cc, double *fc,
        double *oE, double *oN, double *uC,
        double *x, double *w1, double *w2, double *r,
        int *itmax, int *iters,
        double *errtol, double *omega,
        int *iresid, int *iadjoint) {

    int i, j, k, ioff;
    int sz = *nx * *ny * *nz;
    int threads = 512;
    int blocks = (int)ceil(sz/(float)threads);
    
    MAT3(cc, *nx, *ny, *nz);
    MAT3(fc, *nx, *ny, *nz);
    MAT3( x, *nx, *ny, *nz);
    MAT3(w1, *nx, *ny, *nz);
    MAT3(w2, *nx, *ny, *nz);
    MAT3( r, *nx, *ny, *nz);

    MAT3(oE, *nx, *ny, *nz);
    MAT3(oN, *nx, *ny, *nz);
    MAT3(uC, *nx, *ny, *nz);
    MAT3(oC, *nx, *ny, *nz);
    MAT3(write,*nx, *ny, *nz);   

    //intialize cuda arrays
    float *d_x;  HANDLE_ERROR(hipMalloc((void**)&d_x,  sizeof(float) * sz));
    float *d_x2; HANDLE_ERROR(hipMalloc((void**)&d_x2, sizeof(float) * sz));
    float *d_xtemp; HANDLE_ERROR(hipMalloc((void**)&d_xtemp, sizeof(float) * sz));
    float *d_cc; HANDLE_ERROR(hipMalloc((void**)&d_cc, sizeof(float) * sz));
    float *d_fc; HANDLE_ERROR(hipMalloc((void**)&d_fc, sizeof(float) * sz));
    float *d_oC; HANDLE_ERROR(hipMalloc((void**)&d_oC, sizeof(float) * sz));
    float *d_uC; HANDLE_ERROR(hipMalloc((void**)&d_uC, sizeof(float) * sz));
    float *d_oN; HANDLE_ERROR(hipMalloc((void**)&d_oN, sizeof(float) * sz));
    float *d_oE; HANDLE_ERROR(hipMalloc((void**)&d_oE, sizeof(float) * sz));
    
    //float arrays
    float *fx;  fx  = (float*)malloc(sizeof(float)*sz);
    float *ffc; ffc = (float*)malloc(sizeof(float)*sz);
    float *fcc; fcc = (float*)malloc(sizeof(float)*sz);
    float *foC; foC = (float*)malloc(sizeof(float)*sz);
    float *fuC; fuC = (float*)malloc(sizeof(float)*sz);
    float *foN; foN = (float*)malloc(sizeof(float)*sz);
    float *foE; foE = (float*)malloc(sizeof(float)*sz);
    //initialize them to 0
    for(i=0; i<sz; i++){
    	fx[i]  = 0; ffc[i] = 0; fcc[i] = 0;
    	foC[i] = 0; fuC[i] = 0; foN[i] = 0;
    	foE[i] = 0;
    }
    
    MAT3(fx, *nx,*ny,*nz);
    MAT3(ffc, *nx,*ny,*nz);
    MAT3(fcc, *nx,*ny,*nz);
    MAT3(foC, *nx,*ny,*nz);
    MAT3(fuC, *nx,*ny,*nz);
    MAT3(foN, *nx,*ny,*nz);
    MAT3(foE, *nx,*ny,*nz);
    
    for(k=2; k<=*nz-1; k++){
    	for(j=2; j<=*ny-1; j++){
    		for(i=2; i<=*nx-1; i++){
    			VAT3(fx,i,j,k) 	= (float)VAT3(x,i,j,k);
    			VAT3(ffc,i,j,k)	= (float)VAT3(fc,i,j,k);
    			VAT3(fcc,i,j,k) = (float)VAT3(cc,i,j,k);
    			VAT3(foC,i,j,k) = (float)VAT3(oC,i,j,k);
    			VAT3(fuC,i,j,k) = (float)VAT3(uC,i,j,k);
    			VAT3(foN,i,j,k) = (float)VAT3(oN,i,j,k);
    			VAT3(foE,i,j,k) = (float)VAT3(oE,i,j,k);
    		}
    	}
    }
    
    //copy data from host to device
    HANDLE_ERROR(hipMemcpy(d_x,  fx,  sizeof(float)*sz, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_x2, fx,  sizeof(float)*sz, hipMemcpyHostToDevice));
    //HANDLE_ERROR(hipMemcpy(d_xtemp, fx,  sizeof(float)*sz, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_cc, fcc, sizeof(float)*sz, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_fc, ffc, sizeof(float)*sz, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_oC, foC, sizeof(float)*sz, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_uC, fuC, sizeof(float)*sz, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_oN, foN, sizeof(float)*sz, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_oE, foE, sizeof(float)*sz, hipMemcpyHostToDevice));
  
    for (*iters=1; *iters<=*itmax; (*iters)++) {

    	cuTest<<<blocks, threads>>>(d_x, d_x2, d_fc, d_cc, d_oC, d_uC, d_oE, d_oN, sz, *nx, *ny, *nz);
	HANDLE_ERROR(hipGetLastError());
    	float *temp = d_x;
    	d_x = d_x2;
    	d_x2 = temp;
    	
    	
        // Do the red points ***
//        #pragma omp parallel for private(i, j, k, ioff)
//        for (k=2; k<=*nz-1; k++) {
//            for (j=2; j<=*ny-1; j++) {
//                ioff = (1 - *iadjoint) * (    (j + k + 2) % 2)
//                     + (    *iadjoint) * (1 - (j + k + 2) % 2);
//                for (i=2+ioff; i<=*nx-1; i+=2) {
//                    VAT3(x, i, j, k) = (
//                            VAT3(fc,   i,  j,  k)
//                         +  VAT3(oN,   i,   j,   k) * VAT3(x,   i, j+1,   k)
//                         +  VAT3(oN,   i, j-1,   k) * VAT3(x,   i, j-1,   k)
//                         +  VAT3(oE,   i,   j,   k) * VAT3(x, i+1,   j,   k)
//                         +  VAT3(oE, i-1,   j,   k) * VAT3(x, i-1,   j,   k)
//                         + VAT3( uC,   i,   j, k-1) * VAT3(x,   i,   j, k-1)
//                         + VAT3( uC,   i,   j,   k) * VAT3(x,   i,   j, k+1)
//                         ) / (VAT3(oC, i, j, k) + VAT3(cc, i, j, k));
//                }
//            }
//        }
//
//        // Do the black points
//        #pragma omp parallel for private(i, j, k, ioff)
//        for (k=2; k<=*nz-1; k++) {
//            for (j=2; j<=*ny-1; j++) {
//                ioff =   (    *iadjoint) * (    (j + k + 2) % 2 )
//                       + (1 - *iadjoint) * (1 - (j + k + 2) % 2 );
//                for (i=2+ioff;i<=*nx-1; i+=2) {
//                    VAT3(x, i, j, k) = (
//                            VAT3(fc,   i,   j,   k)
//                         +  VAT3(oN,   i,   j,   k) * VAT3(x,   i,j+1,  k)
//                         +  VAT3(oN,   i, j-1,   k) * VAT3(x,   i,j-1,  k)
//                         +  VAT3(oE,   i,   j,   k) * VAT3(x, i+1,  j,  k)
//                         +  VAT3(oE, i-1,   j,   k) * VAT3(x, i-1,  j,  k)
//                         + VAT3( uC,   i,   j, k-1) * VAT3(x,   i,  j,k-1)
//                         + VAT3( uC,   i,   j,   k) * VAT3(x,   i,  j,k+1)
//                         ) / (VAT3(oC, i, j, k) + VAT3(cc, i, j, k));
//                }
//            }
//        }
    }
    
    HANDLE_ERROR(hipDeviceSynchronize());
    //copy data from host to device
    HANDLE_ERROR(hipMemcpy(fx,   d_x, sizeof(float)*sz, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(fcc, d_cc, sizeof(float)*sz, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(ffc, d_fc, sizeof(float)*sz, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(foC, d_oC, sizeof(float)*sz, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(fuC, d_uC, sizeof(float)*sz, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(foN, d_oN, sizeof(float)*sz, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(foE, d_oE, sizeof(float)*sz, hipMemcpyDeviceToHost));
	
    for(k=2; k<=*nz-1; k++){
    	for(j=2; j<=*ny-1; j++){
    		for(i=2; i<=*nx-1; i++){
    			VAT3(x,i,j,k) 	= (double)VAT3(fx,i,j,k);
    			VAT3(fc,i,j,k)	= (double)VAT3(ffc,i,j,k);
    			VAT3(cc,i,j,k) = (double)VAT3(fcc,i,j,k);
    			VAT3(oC,i,j,k) = (double)VAT3(foC,i,j,k);
    			VAT3(uC,i,j,k) = (double)VAT3(fuC,i,j,k);
    			VAT3(oN,i,j,k) = (double)VAT3(foN,i,j,k);
    			VAT3(oE,i,j,k) = (double)VAT3(foE,i,j,k);
    		}
    	}
    }

    //release cuda memory
    HANDLE_ERROR(hipFree(d_x));  HANDLE_ERROR(hipFree(d_cc)); HANDLE_ERROR(hipFree(d_fc));
    HANDLE_ERROR(hipFree(d_oC)); HANDLE_ERROR(hipFree(d_uC)); HANDLE_ERROR(hipFree(d_oE));
    HANDLE_ERROR(hipFree(d_oN)); HANDLE_ERROR(hipFree(d_x2)); 
    
    //release float arrays
    free(fx);  free(ffc); free(fcc);
    free(foC); free(fuC); free(foN);
    free(foE); 

    /*
    FILE *fd;
    fd = fopen("oC.txt", "w+");
    for(i=0; i<sz; i++){
    	fprintf(fd, "(%d):  oC=%f oE=%f oN=%f uC=%f fc=%f cc=%f x=%f write=%d\n", i, oC[i], oE[i], oN[i], uC[i], fc[i], cc[i], x[i], write[i]);
    }
    for(k=2; k<=*nz-1; k++){
    	for(j=2; j<=*ny-1; j++){
    		for(i=2; i<=*nx-1; i++){
    			fprintf(fd, "(index:%d,i=%d,j=%d,k=%d):  oC=%f oE=%f oN=%f uC=%f fc=%f cc=%f x=%f write=%d\n", (k-1)*(*nx)*(*ny) + (j-1)*(*nx) + (i-1),i,j,k,
    			VAT3(oC,i,j,k), VAT3(oE,i,j,k), VAT3(oN,i,j,k), VAT3(uC,i,j,k), VAT3(fc,i,j,k), VAT3(cc,i,j,k), VAT3(x,i,j,k), VAT3(write,i,j,k));
		}
	}
    }
    fclose(fd);
    free(write);
    exit(0);
    */  

    if (*iresid == 1)
        Vmresid7_1s(nx, ny, nz, ipc, rpc, oC, cc, fc, oE, oN, uC, x, r);
}



VPUBLIC void Vgsrb27x(int *nx,int *ny,int *nz,
        int *ipc, double *rpc,
        double  *oC, double  *cc, double  *fc,
        double  *oE, double  *oN, double  *uC, double *oNE, double *oNW,
        double  *uE, double  *uW, double  *uN, double  *uS,
        double *uNE, double *uNW, double *uSE, double *uSW,
        double *x, double *w1, double *w2, double *r,
        int *itmax, int *iters,
        double *errtol, double *omega,
        int *iresid, int *iadjoint) {

    int  i,  j,  k;
    int i1, j1, k1;
    int i2, j2, k2;
    int ioff;
    int istep;

    double tmpO, tmpU, tmpD;

    MAT3( cc, *nx, *ny, *nz);
    MAT3(fc, *nx, *ny, *nz);
    MAT3( x, *nx, *ny, *nz);
    MAT3(w1, *nx, *ny, *nz);
    MAT3(w2, *nx, *ny, *nz);
    MAT3( r, *nx, *ny, *nz);

    MAT3(oE, *nx, *ny, *nz);
    MAT3(oN, *nx, *ny, *nz);
    MAT3(uC, *nx, *ny, *nz);
    MAT3(oC, *nx, *ny, *nz);

    MAT3(oNE, *nx, *ny, *nz);
    MAT3(oNW, *nx, *ny, *nz);

    MAT3( uE, *nx, *ny, *nz);
    MAT3( uW, *nx, *ny, *nz);
    MAT3( uN, *nx, *ny, *nz);
    MAT3( uS, *nx, *ny, *nz);
    MAT3(uNE, *nx, *ny, *nz);
    MAT3(uNW, *nx, *ny, *nz);
    MAT3(uSE, *nx, *ny, *nz);
    MAT3(uSW, *nx, *ny, *nz);

    // Do the gauss-seidel iteration itmax times

    /*
    i1    = (1 - *iadjoint) *   2  + (    *iadjoint) * (*nx - 1);
    i2    = (    *iadjoint) *   2  + (1 - *iadjoint) * (*nx - 1);
    j1    = (1 - *iadjoint) *   2  + (    *iadjoint) * (*ny - 1);
    j2    = (    *iadjoint) *   2  + (1 - *iadjoint) * (*ny - 1);
    k1    = (1 - *iadjoint) *   2  + (    *iadjoint) * (*nz - 1);
    k2    = (    *iadjoint) *   2  + (1 - *iadjoint) * (*nz - 1);
    istep = (    *iadjoint) * (-1) + (1 - *iadjoint) * (1);
    */

    i1 = (1-*iadjoint) * 2 + *iadjoint     * (*nx-1);
    i2 = *iadjoint     * 2 + (1-*iadjoint) * (*nx-1);
    j1 = (1-*iadjoint) * 2 + *iadjoint     * (*ny-1);
    j2 = *iadjoint     * 2 + (1-*iadjoint) * (*ny-1);
    k1 = (1-*iadjoint) * 2 + *iadjoint     * (*nz-1);
    k2 = *iadjoint     * 2 + (1-*iadjoint) * (*nz-1);
    istep = *iadjoint*(-1) + (1-*iadjoint)*(1);

    for (*iters=1; *iters<=*itmax; (*iters)++) {

        //#pragma omp parallel for private(i, j, k, ioff, tmpO, tmpU, tmpD)
        for (k=2; k<=*nz-1; k++) {

            for (j=2; j<=*ny-1; j++) {

                ioff = (1 - *iadjoint) * (    (j + k + 2) % 2)
                     + (    *iadjoint) * (1 - (j + k + 2) % 2);

                for (i=2+ioff; i<=*nx-1; i+=2) {

                    tmpO =
                         + VAT3(  oN,   i,   j,   k) * VAT3(x,   i, j+1,   k)
                         + VAT3(  oN,   i, j-1,   k) * VAT3(x,   i, j-1,   k)
                         + VAT3(  oE,   i,   j,   k) * VAT3(x, i+1,   j,   k)
                         + VAT3(  oE, i-1,   j,   k) * VAT3(x, i-1,   j,   k)
                         + VAT3( oNE,   i,   j,   k) * VAT3(x, i+1, j+1,   k)
                         + VAT3( oNW,   i,   j,   k) * VAT3(x, i-1, j+1,   k)
                         + VAT3( oNW, i+1, j-1,   k) * VAT3(x, i+1, j-1,   k)
                         + VAT3( oNE, i-1, j-1,   k) * VAT3(x, i-1, j-1,   k);

                   tmpU =
                         + VAT3(  uC,   i,   j,   k) * VAT3(x,   i,   j, k+1)
                         + VAT3(  uN,   i,   j,   k) * VAT3(x,   i, j+1, k+1)
                         + VAT3(  uS,   i,   j,   k) * VAT3(x,   i, j-1, k+1)
                         + VAT3(  uE,   i,   j,   k) * VAT3(x, i+1,   j, k+1)
                         + VAT3(  uW,   i,   j,   k) * VAT3(x, i-1,   j, k+1)
                         + VAT3( uNE,   i,   j,   k) * VAT3(x, i+1, j+1, k+1)
                         + VAT3( uNW,   i,   j,   k) * VAT3(x, i-1, j+1, k+1)
                         + VAT3( uSE,   i,   j,   k) * VAT3(x, i+1, j-1, k+1)
                         + VAT3( uSW,   i,   j,   k) * VAT3(x, i-1, j-1, k+1);

                   tmpD =
                         + VAT3(  uC,   i,   j, k-1) * VAT3(x,   i,   j, k-1)
                         + VAT3(  uS,   i, j+1, k-1) * VAT3(x,   i, j+1, k-1)
                         + VAT3(  uN,   i, j-1, k-1) * VAT3(x,   i, j-1, k-1)
                         + VAT3(  uW, i+1,   j, k-1) * VAT3(x, i+1,   j, k-1)
                         + VAT3(  uE, i-1,   j, k-1) * VAT3(x, i-1,   j, k-1)
                         + VAT3( uSW, i+1, j+1, k-1) * VAT3(x, i+1, j+1, k-1)
                         + VAT3( uSE, i-1, j+1, k-1) * VAT3(x, i-1, j+1, k-1)
                         + VAT3( uNW, i+1, j-1, k-1) * VAT3(x, i+1, j-1, k-1)
                         + VAT3( uNE, i-1, j-1, k-1) * VAT3(x, i-1, j-1, k-1);

                       VAT3(x, i,j,k) = (VAT3(fc, i, j, k) + (tmpO + tmpU + tmpD))
                                / (VAT3(oC, i, j, k) + VAT3(cc, i, j, k));

                }
            }
        }

        //#pragma omp parallel for private(i, j, k, ioff, tmpO, tmpU, tmpD)
        for (k=2; k<=*nz-1; k++) {

            for (j=2; j<=*ny-1; j++) {

                ioff = (    *iadjoint) * (    (j + k + 2) % 2)
                     + (1 - *iadjoint) * (1 - (j + k + 2) % 2);

                for (i=2+ioff; i<=*nx-1; i+=2) {

                    tmpO =
                         + VAT3(  oN,   i,   j,   k) * VAT3(x,   i, j+1,   k)
                         + VAT3(  oN,   i, j-1,   k) * VAT3(x,   i, j-1,   k)
                         + VAT3(  oE,   i,   j,   k) * VAT3(x, i+1,   j,   k)
                         + VAT3(  oE, i-1,   j,   k) * VAT3(x, i-1,   j,   k)
                         + VAT3( oNE,   i,   j,   k) * VAT3(x, i+1, j+1,   k)
                         + VAT3( oNW,   i,   j,   k) * VAT3(x, i-1, j+1,   k)
                         + VAT3( oNW, i+1, j-1,   k) * VAT3(x, i+1, j-1,   k)
                         + VAT3( oNE, i-1, j-1,   k) * VAT3(x, i-1, j-1,   k);

                    tmpU =
                         + VAT3(  uC,   i,   j,   k) * VAT3(x,   i,   j, k+1)
                         + VAT3(  uN,   i,   j,   k) * VAT3(x,   i, j+1, k+1)
                         + VAT3(  uS,   i,   j,   k) * VAT3(x,   i, j-1, k+1)
                         + VAT3(  uE,   i,   j,   k) * VAT3(x, i+1,   j, k+1)
                         + VAT3(  uW,   i,   j,   k) * VAT3(x, i-1,   j, k+1)
                         + VAT3( uNE,   i,   j,   k) * VAT3(x, i+1, j+1, k+1)
                         + VAT3( uNW,   i,   j,   k) * VAT3(x, i-1, j+1, k+1)
                         + VAT3( uSE,   i,   j,   k) * VAT3(x, i+1, j-1, k+1)
                         + VAT3( uSW,   i,   j,   k) * VAT3(x, i-1, j-1, k+1);

                   tmpD =
                         + VAT3(  uC,   i,   j, k-1) * VAT3(x,   i,   j, k-1)
                         + VAT3(  uS,   i, j+1, k-1) * VAT3(x,   i, j+1, k-1)
                         + VAT3(  uN,   i, j-1, k-1) * VAT3(x,   i, j-1, k-1)
                         + VAT3(  uW, i+1,   j, k-1) * VAT3(x, i+1,   j, k-1)
                         + VAT3(  uE, i-1,   j, k-1) * VAT3(x, i-1,   j, k-1)
                         + VAT3( uSW, i+1, j+1, k-1) * VAT3(x, i+1, j+1, k-1)
                         + VAT3( uSE, i-1, j+1, k-1) * VAT3(x, i-1, j+1, k-1)
                         + VAT3( uNW, i+1, j-1, k-1) * VAT3(x, i+1, j-1, k-1)
                         + VAT3( uNE, i-1, j-1, k-1) * VAT3(x, i-1, j-1, k-1);

                   VAT3(x, i,j,k) = (VAT3(fc, i, j, k) + (tmpO + tmpU + tmpD))
                            / (VAT3(oC, i, j, k) + VAT3(cc, i, j, k));
                }
            }
        }
    }

    // If specified, return the new residual as well
    if (*iresid == 1)
        Vmresid27_1s(nx, ny, nz,
                     ipc, rpc,
                      oC,  cc,  fc,
                      oE,  oN,  uC,
                     oNE, oNW,
                     uE,   uW,  uN,  uS,
                     uNE, uNW, uSE, uSW,
                       x,   r);
}
